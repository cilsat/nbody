#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define NB 100  // number of bodies
#define ND 3    // number of dimensions
#define DEL 1   // (integer) length of time steps in seconds
#define MAX_VAL_X 100   // maximum positional value
#define MAX_VAL_V 10    // maximum (initial) velocity
#define MAX_VAL_M 10    // maximum mass of body

typedef struct {
    float3 *x;
    float *m;
    float3 *v;
    float3 *F;
    int num;
    float g;
} nbodysys;

nbodysys *init(int n) {
    int i, j;
    nbodysys *s = (nbodysys *)malloc(sizeof(nbodysys));

    srand(time(NULL));
    s->x = (float3 *)malloc(n*sizeof(float3));
    s->m = (float *)malloc(n*sizeof(float));
    s->v = (float *)malloc(n*ND*sizeof(float));
    s->F = (float *)malloc(n*ND*sizeof(float));
    s->num = n;
    s->g = 6.67408*pow(10, -11);

    for (i = 0; i < n; i++) {
        for (j = 0; j < ND; j++) { 
            s->x[i*ND + j] = MAX_VAL_X*((float) rand()/(float) RAND_MAX) - 0.5*MAX_VAL_X;
            s->v[i*ND + j] = MAX_VAL_V*((float) rand()/(float) RAND_MAX);
            s->F[i*ND + j] = 0;
        }
        s->m[i] = MAX_VAL_M*((float) rand()/(float) RAND_MAX);
    }

    return s;
}

void fin(nbodysys *s) {
    free(s->x);
    free(s->m);
    free(s->v);
    free(s);
}

void update(nbodysys *s) {
    int i, j, k;
    float d, f, a;
    float *xr = (float *) malloc(ND*sizeof(float));
    float *a = (float *) malloc(ND*sizeof(float));

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            d = 0;
            for (k = 0; k < ND; k++) {
                xr[k] = s->x[j*ND + k] - s->x[i*ND + k];
                d += powf(xr[k], 2);
            }
            d = powf(d, 0.5);
            f = s->g*(s->m[j] - s->m[i])/d;
            for (k = 0; k < ND; k++) {
                a[k] = 

                

void print(nbodysys *s) {
    int i, j;
    int n = s->num;

    for (i = 0; i < n; i++) {
        for (j = 0; j < ND; j++) { 
            printf("%.3f ", s->x[i*ND + j]);
        }
        printf("\n");
    }
    for (i = 0; i < n; i++) {
        for (j = 0; j < ND; j++) { 
            printf("%.3f ", s->v[i*ND + j]);
        }
        printf("\n");
    }
    for (i = 0; i < n; i++) {
        printf("%.3f ", s->m[i]);
    }
}

int main(int argc, char **argv) {
    nbodysys *sys = init(NB);
    print(sys, NB);
    fin(sys);

    return 0;
}
