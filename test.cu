#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define NB 100  // number of bodies
#define ND 3    // number of dimensions
#define DEL 1   // (integer) length of time steps in seconds
#define MAX_VAL_X 100   // maximum positional value
#define MAX_VAL_V 10    // maximum (initial) velocity
#define MAX_VAL_M 10    // maximum mass of body

typedef struct {
    float *x;
    float *m;
    float *v;
    int num;
    float g;
} nbodysys;

nbodysys *init(int n) {
    int i, j;
    nbodysys *s = (nbodysys *)malloc(sizeof(nbodysys));

    srand(time(NULL));
    s->x = (float *)malloc(n*ND*sizeof(float));
    s->m = (float *)malloc(n*sizeof(float));
    s->v = (float *)malloc(n*ND*sizeof(float));
    s->num = n;
    s->g = 6.6708*pow(10, -11);

    for (i = 0; i < n; i++) {
        for (j = 0; j < ND; j++) { 
            s->x[i*ND + j] = MAX_VAL_X*((float) rand()/(float) RAND_MAX) - 0.5*MAX_VAL_X;
            s->v[i*ND + j] = MAX_VAL_V*((float) rand()/(float) RAND_MAX);
        }
        s->m[i] = MAX_VAL_M*((float) rand()/(float) RAND_MAX);
    }

    return s;
}

void fin(nbodysys *s) {
    free(s->x);
    free(s->m);
    free(s->v);
    free(s);
}

void update(nbodysys *s) {
    int i, j, k;
    float n0, n1;
    float *d = (float *) malloc(ND*sizeof(float));
    float *f = (float *) malloc(n*ND*sizeof(float));

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            for (k = 0; k < ND; k++) {
                d[k] = s->x[j*ND + k] - s->x[i*ND + k];
                

void print(nbodysys *s) {
    int i, j;
    int n = s->num;

    for (i = 0; i < n; i++) {
        for (j = 0; j < ND; j++) { 
            printf("%.3f ", s->x[i*ND + j]);
        }
        printf("\n");
    }
    for (i = 0; i < n; i++) {
        for (j = 0; j < ND; j++) { 
            printf("%.3f ", s->v[i*ND + j]);
        }
        printf("\n");
    }
    for (i = 0; i < n; i++) {
        printf("%.3f ", s->m[i]);
    }
}

int main(int argc, char **argv) {
    nbodysys *sys = init(NB);
    print(sys, NB);
    fin(sys);

    return 0;
}
